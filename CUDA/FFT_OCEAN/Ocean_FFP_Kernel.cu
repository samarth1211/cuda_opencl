#include "hip/hip_runtime.h"
#include"Ocean_kernel.cuh"

int cuda_iDivUp(int a, int b)
{
	return (a+(b-1)) / b;
}

// complex maths functions
__device__ float2 conjugate(float2 arg)
{
	return make_float2(arg.x,arg.y);
}

__device__ float2 complex_exp(float arg)
{
	return make_float2(cosf(arg), sinf(arg));
}

__device__ float2 complex_add(float2 a, float2 b)
{
	return make_float2(a.x+b.x , a.y+b.y);
}

__device__ float2 complex_mult(float2 ab, float2 cd)
{
	return make_float2( (ab.x * cd.x) - (ab.y*cd.y),(ab.x * cd.y) +(ab.y * cd.x));
}

/* 
	generate wave heightfield at time t based on inital heightfiled and dispression relationship
*/
__global__ void generateSpectrumKernel(float2 *h0, float2 *ht, unsigned int in_width, unsigned int out_width, unsigned int out_height, float t, float patchSize)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int in_index = y*in_width + x;
	unsigned int in_mIndex = (out_height -y)*in_width + (out_width-x);// mirrored index
	unsigned int out_index = y*out_width + x;

	// calculate wave vector
	float2 k;
	k.x = (-(int)out_width / 2.0f + x) * (2.0f * M_PI / patchSize);
	k.y = (-(int)out_width / 2.0f + y) * (2.0f * M_PI / patchSize);

	// calculate dispersion w(k)
	float k_len = sqrtf(k.x*k.x + k.y*k.y);
	float w = sqrtf(9.81f * k_len);

	if ((x < out_width)&&(y < out_width))
	{
		float2 h0_k = h0[in_index];
		float2 h0_mk = h0[in_mIndex];

		// output frequency-sdapce complex values
		ht[out_index] = complex_add(complex_mult(h0_k,complex_exp(w*t)),complex_mult(conjugate(h0_mk),complex_exp(-w*t)));
	}
}

// update height map values based on output of fft
__global__ void updateHeightMapKernel(float *heightMap, float2 *ht, unsigned int width)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int i = y*width + x;

	float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;

	heightMap[i] = ht[i].x * sign_correction;
}

__global__ void updateHeightMapKernel_y(float *heightMap, float2 *ht, unsigned int width)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int i = y*width + x;

	float sign_correction = ((x + y) & 0x01) ? -1.0f : 1.0f;

	heightMap[i] = ht[i].y * sign_correction;
}


// generate slope by partial differences in spatial domain
__global__ void calculateSlopeKernel(float *h,float2 *slopeOut,unsigned int width,unsigned int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned int i = y*width + x;

	float2 slope = make_float2(0.0f,0.0f);

	if ((x > 0) && (y > 0) && (x < width - 1) && (y < height - 1))
	{
		slope.x = h[i + 1] - h[i - 1];
		slope.y = h[i + width] - h[i - width];
	}

	slopeOut[i] = slope;
}

void cuda_GenerateSpectrumKernel(float2 *d_h0, float2 *d_ht, unsigned int in_width, unsigned int out_width, unsigned int out_height, float animeTime, float patchSize)
{
	dim3 block(32,32,1);
	dim3 grid(cuda_iDivUp(out_width,block.x), cuda_iDivUp(out_height, block.y),1);

	generateSpectrumKernel <<<grid, block >>>(d_h0,d_ht,in_width,out_width,out_height,animeTime,patchSize);
}

void cuda_UpdateHeightMapKernel(float *d_heightMap, float2 *d_ht, unsigned int width, unsigned int height, bool autoTest)
{
	dim3 block(32, 32, 1);
	dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);
	if (autoTest)
	{
		updateHeightMapKernel_y<<<grid, block>>>(d_heightMap,d_ht,width);
	}
	else
	{
		updateHeightMapKernel<<<grid,block>>>(d_heightMap,d_ht,width);
	}

}

void cuda_CalculateSlopKernel(float * hptr, float2 * slopeOut, unsigned int width, unsigned int height)
{
	dim3 block(32, 32, 1);
	dim3 grid(cuda_iDivUp(width, block.x), cuda_iDivUp(height, block.y), 1);

	calculateSlopeKernel<<<grid,block>>>(hptr,slopeOut,width,height);
}


