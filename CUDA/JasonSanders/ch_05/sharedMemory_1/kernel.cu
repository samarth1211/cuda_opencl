#include "hip/hip_runtime.h"
#include<Windows.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#include<gl\glew.h>
#include<gl\GL.h>

#include<hip/hip_runtime.h>
#include<cuda_gl_interop.h>


#include"vmath.h"


#define		DIM				1024
#define		PI				3.1415926535897932f
#define WIN_WIDTH	800
#define WIN_HEIGHT	600

#pragma comment(lib,"user32.lib")
#pragma comment(lib,"gdi32.lib")
#pragma comment(lib,"glew32.lib")
#pragma comment(lib,"opengl32.lib")
#pragma comment(lib,"cudart.lib")

//using namespace std;
enum CUDAInitErrorCodes
{
	/* min no -10 */
	INIT_CUDA_SETGLDEVICE_FAILED = -21,
	CUDA_STREAM_SYNC_FAILED,
	CUDA_INIT_DESTROY_SURFACE_OBJ_FAILED,
	INIT_CUDA_REGISTER_IMAGE_FAILED,
	INIT_CUDA_REGISTER_BUFFER_FAILED,
	CUDA_INIT_GRAPHICS_MAPPED_ARRAY_FAILED,
	CUDA_INIT_GRAPHICS_MAPPED_RES_FAILED,
	CUDA_INIT_GRAPHICS_MAPPED_RES_POINTER_FAILED,
	CUDA_INIT_GRAPHICS_UNMAP_RES_FAILED,
	CUDA_INIT_GRAPHICS_ERR_1,
	CUDA_INIT_GRAPHICS_ERR_3,
	INIT_CUDA_CHOOSEDEVICE_FAILED = -10,
	CUDA_INIT_ALL_OK = 0,
};


enum InitErrorCodes
{
	INIT_VERTEX_SHADER_COMPILATION_FAILED = -9,
	INIT_FRAGMENT_SHADER_COMPILATION_FAILED,
	INIT_LINK_SHADER_PROGRAM_FAILED,
	INIT_FAIL_GLEW_INIT,
	INIT_FAIL_BRIDGE_CONTEX_SET,
	INIT_FAIL_BRIDGE_CONTEX_CREATION,
	INIT_FAIL_SET_PIXEL_FORMAT,
	INIT_FAIL_NO_PIXEL_FORMAT,
	INIT_FAIL_NO_HDC,
	INIT_ALL_OK,
};

enum attributeBindLocations
{
	SAM_ATTRIBUTE_POSITION = 0,
	SAM_ATTRIBUTE_COLOR,
	SAM_ATTRIBUTE_NORNAL,
	SAM_ATTRIBUTE_TEXTURE0,
};


LRESULT CALLBACK MainWndProc(HWND hwnd, UINT iMsg, WPARAM wParam, LPARAM lParam);

bool g_bWindowActive = false;
HWND g_hwnd = NULL;
HDC  g_hdc = NULL;
HGLRC g_hrc = NULL;

WINDOWPLACEMENT wpPrev;
DWORD dwStyle;
bool g_bFullScreen = false;

FILE *g_pFile = NULL;

// Shaders
//GLuint iVertexShaderObject = 0;
//GLuint iFragmentShaderObject = 0;
GLuint g_ShaderProgramObject = 0;

// All Vertex Buffers
GLuint g_VertexArrayObject = 0;
GLuint g_VertexBufferObject_Position = 0;
GLuint g_VertexBufferObject_TexCoords = 0;

// Uniforms
GLuint g_Uniform_Model_Matrix = 0;
GLuint g_Uniform_View_Matrix = 0;
GLuint g_Uniform_Projection_Matrix = 0;
// sampler
GLuint g_uniform_TextureSampler;

GLuint g_cuda_texture;


// Projection
vmath::mat4 g_PersPectiveProjectionMatrix;

//CUDA Res
hipGraphicsResource *resource = NULL;

float g_fanimate = 0.0f;
bool animation_flag = false;

// cuda kernel
__global__ void normal_kernel(hipSurfaceObject_t target, dim3 texDim, float time)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x + y * blockDim.x * gridDim.x;

	if (x < texDim.x && y < texDim.y)
	{
		// now calculate the value at the position
		float fx = x / (float)DIM - 0.5f;
		float fy = y / (float)DIM - 0.5f;
		float  green = 0.5f + 0.5f * sinf(fabsf(fx * 100 * time) - fabsf(fy * 100 * time));
		float4 data = make_float4(0.0f, green, 0.0f, 1.0f);
		surf2Dwrite(data, target, x * sizeof(float4), y, hipBoundaryModeTrap);

	}
}

__global__ void sharedMem_kernel(hipSurfaceObject_t target, dim3 texDim)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int offset = x + y * blockDim.x * gridDim.x;

	__shared__ float shared[16][16];
	const float period = 128;

	if (x < texDim.x && y < texDim.y)
	{
		shared[threadIdx.x][threadIdx.y] = 255 * (sinf(x * 2.0f * PI /period)+1.0f) * (sinf(y * 2.0f * PI /period)+1.0f) / 4.0f;
		uchar4 data = make_uchar4(0.0f, shared[15-threadIdx.x][15-threadIdx.y], 0.0f,255);
		surf2Dwrite(data, target, x * sizeof(uchar4), y, hipBoundaryModeTrap);
	}
}

int WINAPI WinMain(HINSTANCE hInstance, HINSTANCE hPrevInstance, LPSTR szCmdLine, int iCmdShow)
{
	//int UnInitialize(void);
	int Initialize(void);
	int Update(void);
	void Render(void);

	// Windowing Elelments
	WNDCLASSEX wndclass;
	MSG msg;
	HWND hwnd = NULL;
	TCHAR szClassName[] = TEXT("Sam_OGL");
	RECT windowRect;

	// Game Loop Control
	bool bDone = false;

	// Initialization Status
	int iInitRet = 0;


	SecureZeroMemory((void*)&wndclass, sizeof(wndclass));
	wndclass.cbSize = sizeof(wndclass);
	wndclass.cbClsExtra = 0;
	wndclass.cbWndExtra = 0;
	wndclass.style = CS_HREDRAW | CS_VREDRAW | CS_OWNDC;
	wndclass.lpfnWndProc = MainWndProc;
	wndclass.lpszClassName = szClassName;
	wndclass.lpszMenuName = NULL;
	wndclass.hInstance = hInstance;
	wndclass.hbrBackground = (HBRUSH)GetStockObject(GRAY_BRUSH);
	wndclass.hIcon = LoadIcon(hInstance, IDI_APPLICATION);
	wndclass.hIconSm = LoadIcon(hInstance, IDI_APPLICATION);
	wndclass.hCursor = LoadCursor(hInstance, IDC_ARROW);

	if (!RegisterClassEx(&wndclass))
	{
		MessageBox(NULL, TEXT("Issue...!!!"), TEXT("Could Not RegisterClass() "), MB_OK | MB_ICONERROR);
		exit(EXIT_FAILURE);
	}

	if ((fopen_s(&g_pFile, "SamLogFile.txt", "w+")) == 0)
	{
		fprintf_s(g_pFile, "File Opened Successfully. \n");
	}
	else
	{
		MessageBox(NULL, TEXT("Issue...!!!"), TEXT("Could not open File"), MB_OK | MB_ICONERROR);
		exit(EXIT_FAILURE);
	}

	SecureZeroMemory((void*)&windowRect, sizeof(windowRect));
	windowRect.left = 0;
	windowRect.top = 0;
	windowRect.bottom = WIN_HEIGHT;
	windowRect.right = WIN_WIDTH;
	AdjustWindowRectEx(&windowRect, WS_OVERLAPPEDWINDOW, FALSE, WS_EX_APPWINDOW);

	hwnd = CreateWindowEx(WS_EX_APPWINDOW, szClassName,
		TEXT("First_OpenGL_Window"),
		WS_OVERLAPPEDWINDOW | WS_CLIPCHILDREN | WS_CLIPSIBLINGS | WS_VISIBLE,
		CW_USEDEFAULT, CW_USEDEFAULT,
		windowRect.right - windowRect.left,
		windowRect.bottom - windowRect.top,
		NULL, NULL, hInstance, NULL);

	if (hwnd == NULL)
	{
		MessageBox(NULL, TEXT("Issue...!!!"), TEXT("Could Not CreateWindow() "), MB_OK | MB_ICONERROR);
		exit(EXIT_FAILURE);
	}

	g_hwnd = hwnd;

	iInitRet = Initialize();
	switch (iInitRet)
	{
	case INIT_ALL_OK:
		fprintf_s(g_pFile, "Initialize Complete \n");
		break;
	case INIT_FAIL_NO_HDC:
		fprintf_s(g_pFile, "Failed to Get HDC \n");
		DestroyWindow(hwnd);
		break;
	case INIT_FAIL_NO_PIXEL_FORMAT:
		fprintf_s(g_pFile, "Failed to get PixelFormat \n");
		DestroyWindow(hwnd);
		break;
	case INIT_FAIL_SET_PIXEL_FORMAT:
		fprintf_s(g_pFile, "Failed to set Pixel Format \n");
		DestroyWindow(hwnd);
		break;
	case INIT_FAIL_BRIDGE_CONTEX_CREATION:
		fprintf_s(g_pFile, "Failed to wglCreateContext \n");
		DestroyWindow(hwnd);
		break;
	case INIT_FAIL_BRIDGE_CONTEX_SET:
		fprintf_s(g_pFile, "Failed to wglMakeCurrent \n");
		DestroyWindow(hwnd);
		break;
	case INIT_FAIL_GLEW_INIT:
		fprintf_s(g_pFile, "Failed to glewInit \n");
		DestroyWindow(hwnd);
		break;
	case INIT_LINK_SHADER_PROGRAM_FAILED:
		fprintf_s(g_pFile, "Failed to Link Shader Program Object \n");
		DestroyWindow(hwnd);
		break;
	case INIT_VERTEX_SHADER_COMPILATION_FAILED:
		fprintf_s(g_pFile, "Failed to Compile vertex Shader \n");
		DestroyWindow(hwnd);
		break;
	case INIT_FRAGMENT_SHADER_COMPILATION_FAILED:
		fprintf_s(g_pFile, "Failed to Compile fragment Shader \n");
		DestroyWindow(hwnd);
		break;
	default:
		/*fprintf_s(g_pFile, "Failed UnKnown Reasons \n");
		DestroyWindow(hwnd);*/
		switch (iInitRet)
		{
		case INIT_CUDA_CHOOSEDEVICE_FAILED:
			fprintf_s(g_pFile, "hipChooseDevice Failed  \n");
			DestroyWindow(hwnd);
			break;
			/*default:
			fprintf_s(g_pFile, "CUDA Failed UnKnown Reasons \n");
			DestroyWindow(hwnd);
			break;*/
		}

		// General Failure
		fprintf_s(g_pFile, "Failed UnKnown Reasons \n");
		DestroyWindow(hwnd);
		break;
	}

	ShowWindow(hwnd, SW_SHOWNORMAL);
	SetForegroundWindow(hwnd);
	SetFocus(hwnd);


	while (bDone == false)
	{
		if (PeekMessage(&msg, NULL, 0, 0, PM_REMOVE))
		{
			if (msg.message == WM_QUIT)
			{
				bDone = true;
			}
			else
			{
				TranslateMessage(&msg);
				DispatchMessage(&msg);
			}

		}
		else
		{
			if (g_bWindowActive)
			{
				Update();
			}
			// Show all Animations
			Render();

		}
	}


	//UnInitialize();

	return ((int)msg.wParam);
}


LRESULT CALLBACK MainWndProc(HWND hwnd, UINT iMsg, WPARAM wParam, LPARAM lParam)
{
	int UnInitialize(void);
	void FullScreen(void);
	bool Resize(int, int);
	switch (iMsg)
	{
	case WM_CREATE:
		PostMessage(hwnd, WM_KEYDOWN, (WPARAM)0x46, (LPARAM)NULL);
		break;

	case WM_SETFOCUS:
		g_bWindowActive = true;
		break;

	case WM_KILLFOCUS:
		g_bWindowActive = false;
		break;

	case WM_KEYDOWN:

		switch (LOWORD(wParam))
		{
		case VK_ESCAPE:
			DestroyWindow(hwnd);
			break;

		case 0x46: // 'f' or 'F'
				   //MessageBox(hwnd, TEXT("F is pressed"), TEXT("Status"), MB_OK);
			FullScreen();
			break;

		default:
			break;
		}
		break;

	case WM_SIZE:
		Resize(LOWORD(lParam), HIWORD(lParam));
		break;
	case WM_ERASEBKGND:
		return(0);
		//break;
	case WM_CLOSE:
		DestroyWindow(hwnd);
		break;
	case WM_DESTROY:
		UnInitialize();
		PostQuitMessage(0);
		break;

	default:
		break;
	}

	return (DefWindowProc(hwnd, iMsg, wParam, lParam));
}

int Initialize(void)
{
	

	bool Resize(int, int);
	int iPixelIndex = 0;
	PIXELFORMATDESCRIPTOR pfd;
	hipError_t cuErr;

	// Shader Programs
	GLuint iVertexShaderObject = 0;
	GLuint iFragmentShaderObject = 0;

	GLenum err = NULL; // GLEW Error codes

	SecureZeroMemory(&pfd, sizeof(pfd));
	pfd.nSize = sizeof(pfd);
	pfd.nVersion = 1;
	pfd.dwFlags = PFD_SUPPORT_OPENGL | PFD_DRAW_TO_WINDOW | PFD_DOUBLEBUFFER;
	pfd.iPixelType = PFD_TYPE_RGBA;
	pfd.cColorBits = 32;
	pfd.cRedBits = 8;
	pfd.cGreenBits = 8;
	pfd.cBlueBits = 8;
	pfd.cAlphaBits = 8;

	g_hdc = GetDC(g_hwnd);
	if (g_hdc == NULL)
	{
		return INIT_FAIL_NO_HDC;
	}

	iPixelIndex = ChoosePixelFormat(g_hdc, &pfd);
	if (iPixelIndex == 0)
	{
		return INIT_FAIL_NO_PIXEL_FORMAT;
	}

	if (SetPixelFormat(g_hdc, iPixelIndex, &pfd) == FALSE)
	{
		return INIT_FAIL_SET_PIXEL_FORMAT;
	}

	g_hrc = wglCreateContext(g_hdc);
	if (g_hrc == NULL)
	{
		return INIT_FAIL_BRIDGE_CONTEX_CREATION;
	}

	if (wglMakeCurrent(g_hdc, g_hrc) == FALSE)
	{
		return INIT_FAIL_BRIDGE_CONTEX_SET;
	}

	// Enables Feature Required for Programable Pipeline
	err = glewInit();
	if (err != GLEW_OK)
	{
		return INIT_FAIL_GLEW_INIT;
	}

	// CUDA Initalization

	cuErr = cudaGLSetGLDevice(0); // Default device 0 will share resources with OpenGL
	if (cuErr != hipSuccess)
	{
		return INIT_CUDA_SETGLDEVICE_FAILED;
	}

	// GL information Start
	fprintf_s(g_pFile, "SHADER_INFO : Vendor is : %s\n", glGetString(GL_VENDOR));
	fprintf_s(g_pFile, "SHADER_INFO : Renderer is : %s\n", glGetString(GL_RENDER));
	fprintf_s(g_pFile, "SHADER_INFO : OpenGL Version is : %s\n", glGetString(GL_VERSION));
	fprintf_s(g_pFile, "SHADER_INFO : GLSL Version is : %s\n", glGetString(GL_SHADING_LANGUAGE_VERSION));
	int maxAttachments = 0;
	glGetIntegerv(GL_MAX_COLOR_ATTACHMENTS, &maxAttachments);
	fprintf_s(g_pFile, "SHADER_INFO : GL_MAX_COLOR_ATTACHMENTS is : %d\n", maxAttachments);
	//fprintf_s(g_pFile, "SHADER_INFO : Extention is : %s \n", glGetString(GL_EXTENSIONS));
	// GL information End

	/// Sam : all Shader Code Start
	/*gl_Position =  u_projection_matrix * u_view_matrix * u_model_matrix * vPosition;*/
	/*Vertex Shader Start*/
	iVertexShaderObject = glCreateShader(GL_VERTEX_SHADER);
	const GLchar *vertexShaderSourceCode = "#version 450 core"	\
		"\n" \
		"layout (location = 0)in vec4 vPosition;" \
		"layout (location = 3)in vec2 vTexture0_Coord;" \
		"layout (location = 0)out vec2 out_Texture0_Coord;" \
		"uniform mat4 u_model_matrix,u_view_matrix,u_projection_matrix;" \
		"void main(void)" \
		"{" \
		"	gl_Position =  vPosition;" \
		"	out_Texture0_Coord = vTexture0_Coord;"	\
		"}";

	glShaderSource(iVertexShaderObject, 1, (const GLchar**)&vertexShaderSourceCode, NULL);

	// Compile it
	glCompileShader(iVertexShaderObject);
	GLint iInfoLogLength = 0;
	GLint iShaderCompileStatus = 0;
	GLchar *szInfoLog = NULL;
	glGetShaderiv(iVertexShaderObject, GL_COMPILE_STATUS, &iShaderCompileStatus);
	if (iShaderCompileStatus == GL_FALSE)
	{
		glGetShaderiv(iVertexShaderObject, GL_INFO_LOG_LENGTH, &iInfoLogLength);
		if (iInfoLogLength>0)
		{
			szInfoLog = (GLchar*)malloc(iInfoLogLength * sizeof(GLchar));
			if (szInfoLog != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(iVertexShaderObject, GL_INFO_LOG_LENGTH, &written, szInfoLog);
				fprintf_s(g_pFile, "ERROR : Vertex Shader Compilation Log : %s \n", szInfoLog);
				free(szInfoLog);
				szInfoLog = NULL;
				return INIT_VERTEX_SHADER_COMPILATION_FAILED;
				//DestroyWindow(g_hwnd);
				//exit(EXIT_FAILURE);
			}
		}
	}

	/*Vertex Shader End*/

	/*Fragment Shader Start*/
	iFragmentShaderObject = glCreateShader(GL_FRAGMENT_SHADER);
	const GLchar *fragmentShaderSourceCode = "#version 450 core"	\
		"\n"	\
		"layout (location = 0)in vec2 out_Texture0_Coord;"	\
		"layout (location = 0)out vec4 FragColor;"	\
		"uniform sampler2D u_texture0_sampler;"	\
		"void main(void)"	\
		"{"	\
		"	FragColor = texture(u_texture0_sampler,out_Texture0_Coord);"	\
		"}";

	glShaderSource(iFragmentShaderObject, 1, (const GLchar**)&fragmentShaderSourceCode, NULL);
	glCompileShader(iFragmentShaderObject);
	iInfoLogLength = 0;
	iShaderCompileStatus = 0;
	szInfoLog = NULL;
	glGetShaderiv(iFragmentShaderObject, GL_COMPILE_STATUS, &iShaderCompileStatus);
	if (iShaderCompileStatus == GL_FALSE)
	{
		glGetShaderiv(iFragmentShaderObject, GL_INFO_LOG_LENGTH, &iInfoLogLength);
		if (iInfoLogLength>0)
		{
			szInfoLog = (GLchar*)malloc(iInfoLogLength * sizeof(GLchar));
			if (szInfoLog != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(iFragmentShaderObject, GL_INFO_LOG_LENGTH, &written, szInfoLog);
				fprintf(g_pFile, "ERROR: Fragment Shader Compilation Log : %s \n", szInfoLog);
				free(szInfoLog);
				szInfoLog = NULL;
				return INIT_FRAGMENT_SHADER_COMPILATION_FAILED;
				//DestroyWindow(g_hwnd);
				//exit(EXIT_FAILURE);
			}
		}
	}
	/*Fragment Shader End*/

	/* Shader Program Start */
	g_ShaderProgramObject = glCreateProgram();
	glAttachShader(g_ShaderProgramObject, iVertexShaderObject);
	glAttachShader(g_ShaderProgramObject, iFragmentShaderObject);
	glBindAttribLocation(g_ShaderProgramObject, SAM_ATTRIBUTE_POSITION, "vPosition");
	glBindAttribLocation(g_ShaderProgramObject, SAM_ATTRIBUTE_TEXTURE0, "vTexture0_Coord");
	glLinkProgram(g_ShaderProgramObject);

	GLint iShaderLinkStatus = 0;
	iInfoLogLength = 0;
	glGetProgramiv(g_ShaderProgramObject, GL_LINK_STATUS, &iShaderLinkStatus);
	if (iShaderLinkStatus == GL_FALSE)
	{
		glGetProgramiv(g_ShaderProgramObject, GL_INFO_LOG_LENGTH, &iInfoLogLength);
		if (iInfoLogLength>0)
		{
			szInfoLog = (GLchar*)malloc(iInfoLogLength * sizeof(GLchar));
			if (szInfoLog != NULL)
			{
				GLsizei written;
				glGetShaderInfoLog(g_ShaderProgramObject, GL_INFO_LOG_LENGTH, &written, szInfoLog);
				fprintf_s(g_pFile, "ERROR : Linking Shader Program Objects Failed %s \n", szInfoLog);
				free(szInfoLog);
				szInfoLog = NULL;
				return INIT_LINK_SHADER_PROGRAM_FAILED;
				//DestroyWindow(g_hwnd);
				//exit(EXIT_FAILURE);
			}
		}
	}
	/* Shader Program End */

	/*Setup Uniforms Start*/
	g_Uniform_Model_Matrix = glGetUniformLocation(g_ShaderProgramObject, "u_model_matrix");
	g_Uniform_Projection_Matrix = glGetUniformLocation(g_ShaderProgramObject, "u_projection_matrix");
	g_Uniform_View_Matrix = glGetUniformLocation(g_ShaderProgramObject, "u_view_matrix");
	//g_uniform_TextureSampler = glGetUniformLocation(g_ShaderProgramObject, "u_texture0_sampler");
	/*Setup Uniforms End*/

	/* Fill Buffers Start*/
	
	//// Cube Section Start
	const GLfloat squareVertices[] = {
		-1.0f, 1.0f, 0.0f,
		-1.0f, -1.0f, 0.0f,
		1.0f, -1.0f, 0.0f,
		1.0f, 1.0f, 0.0f
	};

	const GLfloat squareTexCords[] =
	{
		0.0f, 1.0f,
		0.0f, 0.0f,
		1.0f,0.0f,
		1.0f,1.0f
	};

	glGenVertexArrays(1, &g_VertexArrayObject);//VAO
	glBindVertexArray(g_VertexArrayObject);

	glGenBuffers(1, &g_VertexBufferObject_Position);// vbo position
	glBindBuffer(GL_ARRAY_BUFFER, g_VertexBufferObject_Position);
	glBufferData(GL_ARRAY_BUFFER, sizeof(squareVertices), squareVertices, GL_STATIC_DRAW);

	glVertexAttribPointer(SAM_ATTRIBUTE_POSITION, 3, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(SAM_ATTRIBUTE_POSITION);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	glGenBuffers(1, &g_VertexBufferObject_TexCoords); // vbo texcoords
	glBindBuffer(GL_ARRAY_BUFFER, g_VertexBufferObject_TexCoords);
	glBufferData(GL_ARRAY_BUFFER, sizeof(squareTexCords), squareTexCords, GL_STATIC_DRAW);

	glVertexAttribPointer(SAM_ATTRIBUTE_TEXTURE0, 2, GL_FLOAT, GL_FALSE, 0, NULL);
	glEnableVertexAttribArray(SAM_ATTRIBUTE_TEXTURE0);
	glBindBuffer(GL_ARRAY_BUFFER, 0);

	glBindVertexArray(0);
	/* Fill Buffers End*/

	// Generate texture for working with cuda
	glGenTextures(1, &g_cuda_texture);
	glBindTexture(GL_TEXTURE_2D, g_cuda_texture);

	// Texture parameters
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_LINEAR);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	// give texture some storage
	glTexImage2D(GL_TEXTURE_2D,0,GL_RGBA, DIM, DIM,0,GL_RGBA,GL_UNSIGNED_INT,NULL);
	glBindTexture(GL_TEXTURE_2D, 0);
	/// Sam : all Shader Code End

	glEnable(GL_TEXTURE_2D);

	glShadeModel(GL_SMOOTH);
	glClearDepth(1.0f);
	glEnable(GL_DEPTH_TEST);
	glDepthFunc(GL_LEQUAL);
	glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);
	//glEnable(GL_CULL_FACE);

	glClearColor(0.125f, 0.125f, 0.125f, 1.0f);

	
	g_PersPectiveProjectionMatrix = vmath::mat4::identity();

	/// Register With CUDA  Start
	// last param as "hipGraphicsRegisterFlagsSurfaceLoadStore"
	cuErr = hipGraphicsGLRegisterImage(&resource, g_cuda_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
	if (cuErr!=hipSuccess)
	{
		fprintf_s(g_pFile, "CUDA ERROR : hipGraphicsGLRegisterImage failed at line %d\n",__LINE__);
		return INIT_CUDA_REGISTER_IMAGE_FAILED;
	}

	/// Register With CUDA  Start
	Resize(WIN_WIDTH, WIN_HEIGHT);

	return INIT_ALL_OK;
}

int Update(void)
{

	if (animation_flag)
	{
		g_fanimate = g_fanimate + 0.005f;
		if ((g_fanimate >1.0f))
		{
			animation_flag = false;
		}
	}
	else
	{
		g_fanimate = g_fanimate - 0.005f;
		if ((g_fanimate <0.0f))
		{
			animation_flag = true;
		}
	}


	//uchar4 *devPtr = NULL;
	size_t size;
	hipError_t status;
	hipArray_t cudaWriteArray;

	status = hipGraphicsMapResources(1, &resource, 0);
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile,"IN Update()  hipGraphicsMapResources failed...!! \n");
		return CUDA_INIT_GRAPHICS_MAPPED_RES_FAILED;
	}

	status = hipGraphicsSubResourceGetMappedArray(&cudaWriteArray,resource,0,0);
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN Update()  hipGraphicsSubResourceGetMappedArray failed...!! \n");
		return CUDA_INIT_GRAPHICS_MAPPED_ARRAY_FAILED;
	}

	// Prepare a Surface object for cuda
	hipResourceDesc writeDescriptor;
	ZeroMemory((void**)&writeDescriptor,sizeof(writeDescriptor));
	writeDescriptor.resType = hipResourceTypeArray;
	writeDescriptor.res.array.array = cudaWriteArray;

	hipSurfaceObject_t writeSurface;
	status = hipCreateSurfaceObject(&writeSurface, &writeDescriptor);
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN Update()  hipCreateSurfaceObject failed...!! \n");
		return CUDA_INIT_GRAPHICS_MAPPED_ARRAY_FAILED;
	}

	
	// After successfully creating surface object write to the texture using kernel
	// dim3 thread(32,32);
	// dim3 block(DIM/ thread.x, DIM / thread.y);
	//normal_kernel << <block, thread >> >(writeSurface, dim3(DIM, DIM), g_fanimate);
	dim3 thread(16,16);
	dim3 block(DIM/ thread.x, DIM / thread.y);
	sharedMem_kernel<< <block, thread >> >(writeSurface, dim3(DIM, DIM));

	status = hipGetLastError();
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN Update() Kernel failed : %s \n", hipGetErrorString(status));
	}

	/*status = hipDeviceSynchronize();
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN Update() hipDeviceSynchronize failed...!! \n");
		return CUDA_INIT_DESTROY_SURFACE_OBJ_FAILED;
	}*/

	status = hipDestroySurfaceObject(writeSurface);
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN Update() hipDestroySurfaceObject failed...!! \n");
		return CUDA_INIT_DESTROY_SURFACE_OBJ_FAILED;
	}

	status = hipGraphicsUnmapResources(1, &resource, 0);
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN Update() hipGraphicsUnmapResources failed...!! \n");
		return CUDA_INIT_GRAPHICS_UNMAP_RES_FAILED;
	}

	status = hipStreamSynchronize(0);
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN Update() hipStreamSynchronize failed...!! \n");
		return CUDA_INIT_GRAPHICS_UNMAP_RES_FAILED;
	}

	return INIT_ALL_OK;
}

void Render(void)
{
	glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

	vmath::mat4 modelMatrix = vmath::mat4::identity();
	vmath::mat4 viewMatrix = vmath::mat4::identity();

	glUseProgram(g_ShaderProgramObject);

	modelMatrix = vmath::translate(0.0f, 0.0f, -3.0f);

	glUniformMatrix4fv(g_Uniform_Model_Matrix, 1, GL_FALSE, modelMatrix);
	glUniformMatrix4fv(g_Uniform_View_Matrix, 1, GL_FALSE, viewMatrix);
	glUniformMatrix4fv(g_Uniform_Projection_Matrix, 1, GL_FALSE, g_PersPectiveProjectionMatrix);

	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, g_cuda_texture);
	//glUniform1i(g_uniform_TextureSampler, 0);

	glBindVertexArray(g_VertexArrayObject);
	glDrawArrays(GL_TRIANGLE_FAN, 0, 4);
	glBindVertexArray(0);

	glUseProgram(0);

	SwapBuffers(g_hdc);
}

void FullScreen(void)
{
	MONITORINFO mi = { sizeof(mi) };
	dwStyle = GetWindowLong(g_hwnd, GWL_STYLE);
	if (g_bFullScreen == false)
	{
		if (dwStyle & WS_OVERLAPPEDWINDOW)
		{
			if (GetWindowPlacement(g_hwnd, &wpPrev) && GetMonitorInfo(MonitorFromWindow(g_hwnd, MONITORINFOF_PRIMARY), &mi))
			{
				SetWindowLong(g_hwnd, GWL_STYLE, dwStyle & ~WS_OVERLAPPEDWINDOW);
				SetWindowPos(g_hwnd, HWND_TOP,
					mi.rcMonitor.left, mi.rcMonitor.top,
					mi.rcMonitor.right - mi.rcMonitor.left,
					mi.rcMonitor.bottom - mi.rcMonitor.top, SWP_NOZORDER | SWP_FRAMECHANGED);
			}
		}
		ShowCursor(FALSE);
		g_bFullScreen = true;
	}
	else
	{
		SetWindowLong(g_hwnd, GWL_STYLE, dwStyle | WS_OVERLAPPEDWINDOW);
		SetWindowPlacement(g_hwnd, &wpPrev);
		SetWindowPos(g_hwnd, HWND_TOP, 0, 0, 0, 0, SWP_NOOWNERZORDER | SWP_NOZORDER | SWP_FRAMECHANGED | SWP_NOMOVE | SWP_NOSIZE);
		ShowCursor(TRUE);
		g_bFullScreen = false;
	}
}

bool Resize(int iWidth, int iHeight)
{
	if (iHeight <= 0)
	{
		iHeight = 1;
	}

	glViewport(0, 0, (GLsizei)iWidth, (GLsizei)iHeight);

	g_PersPectiveProjectionMatrix = vmath::perspective(45.0f, (float)iWidth / (float)iHeight, 0.1f, 100.0f);

	return true;
}

int UnInitialize(void)
{
	hipError_t status;

	if (g_bFullScreen == true)
	{
		SetWindowLong(g_hwnd, GWL_STYLE, dwStyle | WS_OVERLAPPEDWINDOW);
		SetWindowPlacement(g_hwnd, &wpPrev);
		SetWindowPos(g_hwnd, HWND_TOP, 0, 0, 0, 0, SWP_NOOWNERZORDER | SWP_NOZORDER | SWP_FRAMECHANGED | SWP_NOMOVE | SWP_NOSIZE);
		ShowCursor(TRUE);
		g_bFullScreen = false;
	}

	// Uninitalize CUDA objects
	status = hipGraphicsUnmapResources(1, &resource, 0);
	if (status != hipSuccess)
	{
		fprintf_s(g_pFile, "IN UnInitialize() hipGraphicsUnmapResources failed...!! \n");
		hipDeviceReset();
		return CUDA_INIT_GRAPHICS_UNMAP_RES_FAILED;
	}




	if (g_VertexBufferObject_TexCoords)
	{
		glDeleteBuffers(1, &g_VertexBufferObject_TexCoords);
		g_VertexBufferObject_TexCoords = NULL;
	}

	if (g_VertexBufferObject_Position)
	{
		glDeleteBuffers(1, &g_VertexBufferObject_Position);
		g_VertexBufferObject_Position = NULL;
	}

	if (g_VertexArrayObject)
	{
		glDeleteVertexArrays(1, &g_VertexArrayObject);
		g_VertexArrayObject = NULL;
	}

	glUseProgram(0);
	/*
	glDetachShader(g_ShaderProgramObject, iVertexShaderObject);
	glDetachShader(g_ShaderProgramObject, iFragmentShaderObject);

	if (iFragmentShaderObject)
	{
	glDeleteShader(iFragmentShaderObject);
	iFragmentShaderObject = 0;
	}

	if (iVertexShaderObject)
	{
	glDeleteShader(iVertexShaderObject);
	iVertexShaderObject = 0;
	}

	if (g_ShaderProgramObject)
	{
	glDeleteProgram(g_ShaderProgramObject);
	g_ShaderProgramObject = NULL;
	}*/

	if (g_ShaderProgramObject)
	{
		GLsizei iShaderCount;
		GLsizei iShaderNumber;


		glUseProgram(g_ShaderProgramObject);
		glGetProgramiv(g_ShaderProgramObject, GL_ATTACHED_SHADERS, &iShaderCount);
		GLuint *pShaders = (GLuint*)calloc(iShaderCount, sizeof(GLuint));

		if (pShaders)
		{
			glGetAttachedShaders(g_ShaderProgramObject, iShaderCount, &iShaderCount, pShaders);
			for (iShaderNumber = 0; iShaderNumber < iShaderCount; iShaderNumber++)
			{
				glDetachShader(g_ShaderProgramObject, pShaders[iShaderNumber]);
				glDeleteShader(pShaders[iShaderNumber]);
				pShaders[iShaderNumber] = 0;
			}
			free(pShaders);
			pShaders = NULL;
		}

		glUseProgram(0);

		glDeleteProgram(g_ShaderProgramObject);
		g_ShaderProgramObject = NULL;

	}

	if (g_cuda_texture)
	{
		glDeleteTextures(1,&g_cuda_texture);
		g_cuda_texture = 0;
	}

	if (wglGetCurrentContext() == g_hrc)
	{
		wglMakeCurrent(NULL, NULL);
	}

	if (g_hrc)
	{
		wglDeleteContext(g_hrc);
		g_hrc = NULL;
	}

	if (g_hdc)
	{
		ReleaseDC(g_hwnd, g_hdc);
		g_hdc = NULL;
	}


	if (g_pFile)
	{
		fprintf_s(g_pFile, "Closing File \n");
		fclose(g_pFile);
		g_pFile = NULL;
	}
	return 0;
}
